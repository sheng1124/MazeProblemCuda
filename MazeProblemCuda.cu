#include "hip/hip_runtime.h"
#include "book.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define MAX_STEPS 32


void readFile();
int cpuPathTest(int limitSteps, unsigned long long *tid);
void printMaze();
void printPath(unsigned long long tid, int steps);
void printPathMaze(unsigned long long tid, int steps);
void setTime0();
void getExeTime();

struct Maze
{
	char maze[99][99];
	int rows, cols, s_x, s_y, e_x, e_y;
};

struct Maze maze;
FILE *MAZE;
struct timespec t_start, t_end;
double elapsedTime;

const int threadsPerBlock = 1024;
const int blocksPerGrid = 1024;

__global__ void testPath(int *limitSteps, struct Maze *maze, int *workDone , unsigned long long *path)
{
	unsigned long long tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned long long bias = blockDim.x * gridDim.x;
	unsigned long long maxRoute = 0xffffffffffffffff - (bias - 1);//max length of path
	maxRoute >>= (MAX_STEPS - *limitSteps) * 2;//(32 - 1) * 2 = 62 =>0~011 only 3 steps : right up left 
	
	while(tid <= maxRoute)
	{
		if(*workDone) break;
		
		int x = maze->s_x, y = maze->s_y;
		unsigned long long temp = tid;
		
		int i = *limitSteps;
		int steps = 0;
		do
        {
            //GetMoveDirection
            steps++;
            int direction = temp & 3;//mask
            temp >>= 2;
            //Move
            switch(direction)
            {
            case 0 :
                x += 1;
                break;
            case 1 :
                y -= 1;
                break;
            case 2 :
                x -= 1;
                break;
            case 3 :
                y += 1;
                break;
            }
            //if at Target, print path ,else keep going, if no way then break
            if(maze->maze[y][x] == '$')
            {
				*workDone = 1;
				*path = tid;
                break;
            }
            else if(maze->maze[y][x] != '.')
            {
                break;
            }
        }
        while(i--);
		
		tid += bias;
	}
}


int main()
{
	printf("GPU Version\n");
    readFile();
    printMaze();
	
	// allocate the memory on the GPU
	struct Maze *maze_ptr;
	HANDLE_ERROR(hipMalloc((void**)&maze_ptr, sizeof(struct Maze)));
	int *limitSteps;
	HANDLE_ERROR(hipMalloc((void**)&limitSteps, sizeof(int)));
	int *workDone;
	HANDLE_ERROR(hipMalloc((void**)&workDone, sizeof(int)));
	unsigned long long *path;
	HANDLE_ERROR(hipMalloc((void**)&path, sizeof(unsigned long long)));
	// copy to the GPU
	HANDLE_ERROR(hipMemcpy( maze_ptr, &maze, sizeof(struct Maze), hipMemcpyHostToDevice ));
	
	
	// Get start time event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
	
	// start
	int *isdone = (int*)malloc( sizeof(int) );
	unsigned long long *path_ptr = (unsigned long long*)malloc( sizeof(unsigned long long) );
	int i;

	for(i = 1;i < MAX_STEPS; i++)
	{
		*isdone = 0;
		// copy to the GPU for every turn
		HANDLE_ERROR(hipMemcpy( limitSteps, &i, sizeof(int), hipMemcpyHostToDevice ));
		HANDLE_ERROR(hipMemcpy( workDone, isdone, sizeof(int), hipMemcpyHostToDevice ));
		testPath<<<blocksPerGrid,threadsPerBlock>>>(limitSteps, maze_ptr, workDone, path);
		// copy back from the GPU to the CPU
		
		HANDLE_ERROR( hipMemcpy( isdone, workDone, sizeof(int), hipMemcpyDeviceToHost ) );
		HANDLE_ERROR( hipMemcpy( path_ptr, path, sizeof(unsigned long long), hipMemcpyDeviceToHost ) );
		
		if(*isdone)
		{
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop); 
			break;
		}
		
		// Get stop time event    
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop); 
	}
	printf("search length: %d\n", i);
	printPath(*path_ptr, i);
	printPathMaze(*path_ptr, i);
	//check cuda error
    hipError_t status = hipGetLastError();
    if ( hipSuccess != status ){
        fprintf(stderr, "Error: %s\n", hipGetErrorString(status));
        exit(1) ;
    }
	
    // Compute execution time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("GPU time: %13f msec\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
	
	printf("\nCPU Version\n");
	readFile();
	

	unsigned long long tid;
	setTime0();
	for(i = 1; i < MAX_STEPS; i++)
	{
		if(cpuPathTest(i, &tid)) break;
	}
	getExeTime();
	printf("search length: %d\n", i);
	printPath(tid, i);
	printPathMaze(tid, i);
    fclose(MAZE);
    return 0;
}

int cpuPathTest(int limitSteps, unsigned long long *tid)
{
	*tid = 0;
	unsigned long long bias = 1;
	unsigned long long maxRoute = 0xffffffffffffffff - (bias - 1);//max length of path
	maxRoute >>= (MAX_STEPS - limitSteps) * 2;//(32 - 1) * 2 = 62 =>0~011 only 3 steps : right up left 
	
	while(*tid <= maxRoute)
	{
		int x = maze.s_x, y = maze.s_y;
		unsigned long long temp = *tid;
		
		int i = limitSteps;
		int steps = 0;
		do
        {
            //GetMoveDirection
            steps++;
            int direction = temp & 3;//mask
            temp >>= 2;
            //Move
            switch(direction)
            {
            case 0 :
                x += 1;
                break;
            case 1 :
                y -= 1;
                break;
            case 2 :
                x -= 1;
                break;
            case 3 :
                y += 1;
                break;
            }
            //if at Target, print path ,else keep going, if no way then break
            if(maze.maze[y][x] == '$')
            {
				return 1; //this moment tid not change
            }
            else if(maze.maze[y][x] != '.')
            {
                break;
            }
        }
        while(i--);
		
		*tid += bias;
	}
	return 0;
}




void setTime0()
{
    clock_gettime(CLOCK_REALTIME, &t_start);
}

void getExeTime()
{
    clock_gettime(CLOCK_REALTIME, &t_end);
	elapsedTime = (t_end.tv_sec - t_start.tv_sec) * 1000.0;
	elapsedTime += (t_end.tv_nsec - t_start.tv_nsec) / 1000000.0;
	printf("CPU time: %13f msec\n", elapsedTime);
}

void printPath(unsigned long long tid, int steps)
{
	int x = maze.s_x, y = maze.s_y;
    printf("path = %llx\n", tid);
    do
    {
        int direction = tid & 3;//mask
        tid >>= 2;
        switch(direction)
        {
        case 0:
			x += 1;
            printf("> ");
            break;
        case 1:
			y -= 1;
            printf("^ ");
            break;
        case 2:
			x -= 1;
            printf("< ");
            break;
        case 3:
			y += 1;
            printf("v ");
            break;
        }
		
		if(maze.maze[y][x] == '$')
        {
			break;
        }
    }
    while(steps--);
    printf("\n");
    return;
}

void printPathMaze(unsigned long long tid, int steps)
{
	int x = maze.s_x, y = maze.s_y;
    do
    {
        int direction = tid & 3;//mask
        tid >>= 2;
        switch(direction)
        {
        case 0:				
			maze.maze[y][x] = '>';
			x += 1;
            break;
        case 1:			
			maze.maze[y][x] = '^';
			y -= 1;
            break;
        case 2:			
			maze.maze[y][x] = '<';
			x -= 1;
            break;
        case 3:
			maze.maze[y][x] = 'v';
			y += 1;
            break;
        }
		
		if(maze.maze[y][x] == '$')
        {
			break;
        }
		
    }
    while(steps--);
	
	printMaze();

    return;
}


void readFile()
{
    MAZE = fopen("maze.txt", "r");
    fscanf(MAZE, "%d %d", &maze.rows, &maze.cols);
    fgetc(MAZE);
    printf("rows = %d, cols = %d\n", maze.rows, maze.cols);

    int i = 0, j = 0;
    char c = fgetc(MAZE);
    while((c = fgetc(MAZE)) != EOF)
    {
        if(c == '\n')
        {
            i++;
            j = 0;
        }
        else
        {
            maze.maze[i][j] = c;
            if(c == '*')
            {
                maze.s_x = j;
                maze.s_y = i;
            }
            else if(c == '$')
            {
                maze.e_x = j;
                maze.e_y = i;
            }
            j++;
        }
    }
    return;
}

void printMaze()
{
    printf("start = (%d %d) \nend=(%d %d) \n", maze.s_x, maze.s_y, maze.e_x, maze.e_y);
    int i,j;
    for(i = 0; i < maze.rows; i++)
    {
        for(j = 0; j < maze.cols; j++)
        {
            printf("%c ", maze.maze[i][j]);
        }
        printf("\n");
    }
    return;
}


/*
8 13
#############
#*#...#...#.#
#.#.#.#.#.#.#
#.#.#.#.#.#.#
#...#.......#
##########.##
#$..........#
#############
0xAAAAAF003F05400
*/
